#include "hip/hip_runtime.h"
/**
* This file contains the code of the edge detection example
* program.
* Author: Irina Reshodko
* For GPU Skill Pill @ OIST, Nov. 2017
*/
#include <iostream>
#include <string>
#include "CIMG/CImg.h"
#include <hip/hip_runtime.h>

using namespace cimg_library;

__global__ void stubKernel(const unsigned char * dInData, unsigned char * dOutData, int dataSize)
{
  // 2D grid with 2D blocks
  int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int indx = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
  if (indx >= dataSize)
    return;
  dOutData[indx] = dInDatam[indx] * (unsigned char)((threadIdx.x + threadIdx.y * blockDim.x)
                    /(float)(blockDim.x*blockDim.y));
  return;
}

int main(int argc, char *argv[])
{
  // Get file name from the command line
  std::string fileName("images/thebrain.png");
  if (argc > 1)
    fileName = std::string(argv[2]);
  std::cout << fileName;
  // Load the image
  CImg<unsigned char> image(fileName);
  // Get image dimensions
  int szX = image.width();
  int szY = image.height();
  // Extract the channels
  const unsigned char * redChannel = &image(0, 0, 0, 0);
  const unsigned char * greenChannel = &image(0, 0, 0, 1);
  const unsigned char * blueChannel = &image(0, 0, 0, 2);
  // Create the output image as a 3-channel image of the same size as input
  CImg<unsigned char> outImage(image, false);
  // Extract the channels form the output image
  unsigned char * outRedCh = &outImage(0, 0, 0, 0);
  unsigned char * outGreenCh = &outImage(0, 0, 0, 1);
  unsigned char * outBlueCh = &outImage(0, 0, 0, 2);
  // Test if it works
  outRedCh[20] = 200;
  outGreenCh[30] = 200;
  outBlueCh[10] = 200;
  outImage.save_png("images/test.png");
  // Get cuda properties
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  // Maximum threads per block on this device
  int maxThreads = prop.maxThreadsPerBlock;
  std::cout << "Maximum threads per block: " << maxThreads;
  // Allocate memory on the GPU
  int dataSz = szX * szY * sizeof(unsigned char);
  unsigned char * d_redCh = null;
  unsigned char * d_greenCh = null;
  unsigned char * d_blueCh = null;
  checkCudaErrors(hipMalloc(&d_redCh, dataSz));
  checkCudaErrors(hipMalloc(&d_greenCh, dataSz));
  checkCudaErrors(hipMalloc(&d_blueCh, dataSz));
  unsigned char * d_outRedCh = null;
  unsigned char * d_outGreenCh = null;
  unsigned char * d_outBlueCh = null;
  checkCudaErrors(hipMalloc(&d_outRedCh, dataSz));
  checkCudaErrors(hipMalloc(&d_outGreenCh, dataSz));
  checkCudaErrors(hipMalloc(&d_outBlueCh, dataSz));
  // Copy image data to GPU
  checkCudaErrors(hipMemcpy(d_redCh, redChannel, dataSz, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_greenCh, greenChannel, dataSz, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_blueCh, blueChannel, dataSz, hipMemcpyHostToDevice));
  // Set reasonable block size (i.e., number of threads per block)
  const dim3 blockSize((int) sqrt(maxThreads), (int) sqrt(maxThreads), 1);
  // Compute correct grid size (i.e., number of blocks per kernel launch)
  // from the image size and and block size.
  const dim3 gridSize(int(szX/blockSize.x)+1, int(szY/blockSize.y)+1, 1);
  // Run kernel
  stubKernel<<gridSize, blockSize>>(d_redCh, d_outRedCh, szX * szY);
  stubKernel<<gridSize, blockSize>>(d_greenCh, d_outGreenCh, szX * szY);
  stubKernel<<gridSize, blockSize>>(d_blueCh, d_outBlueCh, szX * szY);
  // Copy results back to CPU
  checkCudaErrors(hipMemcpy(outRedCh, d_outRedCh, dataSz, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(outGreenCh, d_outGreenCh, dataSz, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(outBlueCh, d_outBlueCh, dataSz, hipMemcpyDeviceToHost));
  // Clean up GPU memory
  checkCudaErrors(hipFree(d_redCh));
  checkCudaErrors(hipFree(d_greenCh));
  checkCudaErrors(hipFree(d_blueCh));
  checkCudaErrors(hipFree(d_outRedCh));
  checkCudaErrors(hipFree(d_outGreenCh));
  checkCudaErrors(hipFree(d_outBlueCh));
  // Save resulting image
  outImage.save_png("images/out.png");
}
