/*------------check.cu------------------------------------------------------//
*
* Purpose: This is a simple cuda file for checking your gpu works
*
*          It prints 0 -> 63
*
*-----------------------------------------------------------------------------*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__ void findID(double *a, int n){

    // First we need to find our global threadID
    int id = blockIdx.x*blockDim.x + threadIdx.x;

    // Make sure we are not out of range
    if (id < n){
        a[id] = id;
    }
}

int main(){

    // size of vectors
    int n = 64;

    // Host vectors
    double *h_a;

    // Device vectors
    double *d_a;

    // allocating space on host and device
    h_a = (double*)malloc(sizeof(double)*n);

    // Allocating space on GPU
    hipMalloc(&d_a, sizeof(double)*n);

    // Creating blocks and grid ints
    int threads, grid;

    threads = 64;
    grid = (int)ceil((float)n/threads);

    findID<<<grid, threads>>>(d_a, n);

    // Now to copy c back
    hipMemcpy(h_a, d_a, sizeof(double)*n, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; ++i){
        std::cout << h_a[i] << '\n';
    }

    // Release memory
    hipFree(d_a);

    free(h_a);
}
